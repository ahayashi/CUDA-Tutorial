#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello()
{
    printf("Hello\n");
}

int main()
{
    hello<<<1,1>>>();    
    hipDeviceSynchronize();
    
    return 0;
}    
